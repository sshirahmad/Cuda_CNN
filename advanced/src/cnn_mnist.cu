#include <../lib/cnnlayer.h>
#include <../lib/augmentations.h>
#include <../lib/utils.h>
#include <random>

std::tuple<std::vector<float*>, int, int, int, std::vector<std::string>> read_images(const fs::path& directory) {
    std::vector<float*> images;
    std::vector<std::string> basenames;
    int width = 0;
    int height = 0;
    int channels = 0;

    for (const auto& entry : fs::directory_iterator(directory)) {

        if (entry.is_regular_file() && entry.path().extension() == ".png") {
            // Read the image in grayscale or unchanged
            cv::Mat img = cv::imread(entry.path().string(), cv::IMREAD_UNCHANGED);
            std::string basename = entry.path().stem().string(); 

            if (!img.empty()) {
                width = img.cols;
                height = img.rows;
                channels = img.channels();

                // Allocate memory for float image
                size_t img_size = width * height * channels;
                float* image = AllocateHostMemory<float>(img_size * sizeof(float), "pinned");

                // Convert and copy image data to float
                for (size_t i = 0; i < img_size; ++i) {
                    image[i] = static_cast<float>(img.data[i]);
                }

                images.push_back(image);
                basenames.push_back(basename);

            } else {
                std::cerr << "Failed to load image: " << entry.path() << std::endl;
            }
        } else {
            std::cerr << "Entry is not a regular file or not a PNG: " << entry.path() << std::endl;
        }
    }

    return {images, width, height, channels, basenames}; 
}


std::tuple<std::string, int, int> parseArguments(int argc, char* argv[]) {
    // Initialize default values
    std::string directory = "../data/train/mnist_images";
    int dstWidth = 320;
    int dstHeight = 240;

    // Iterate through command-line arguments
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];

        // Check for the directory flag
        if (arg == "-d" && i + 1 < argc) {
            directory = argv[++i];
        }

        // Check for the width flag
        else if (arg == "-w" && i + 1 < argc) {
            try {
                dstWidth = std::stoi(argv[++i]);
            } catch (const std::invalid_argument& e) {
                std::cerr << "Invalid width value provided. Using default value 320." << std::endl;
            }
        }
        // Check for the height flag
        else if (arg == "-h" && i + 1 < argc) {
            try {
                dstHeight = std::stoi(argv[++i]);
            } catch (const std::invalid_argument& e) {
                std::cerr << "Invalid height value provided. Using default value 240." << std::endl;
            }
        }
    }

    std::cout << "Data Path: " << directory << std::endl;
    std::cout << "Width: " << dstWidth << std::endl;
    std::cout << "Height: " << dstHeight << std::endl;


    return {directory, dstWidth, dstHeight};
}


__host__ void convertToUnsignedChar(const float* input, unsigned char* output, int size) {

    float minRange = *std::min_element(input, input + size);
    float maxRange = *std::max_element(input, input + size);

    std::cout << minRange << std::endl;
    std::cout << maxRange << std::endl;

    if (maxRange == minRange) {
        std::fill(output, output + size, 0);  
    } else {
        for (int i = 0; i < size; ++i) {
            unsigned char scaledValue = static_cast<unsigned char>(255.0f * (input[i] - minRange) / (maxRange - minRange));
            output[i] = scaledValue;
        }
    }
}


__host__ void save_image(int outputWidth, int outputHeight, const float* convImage, int numChannels, std::string filename){

    // Calculate size of image
    int output_size = outputWidth * outputHeight * numChannels;
    size_t conv_size = output_size * sizeof(float);

    // Allocate dynamic memory to host image with flot and unsigned char types
    float* h_conv_image = new float[output_size]; // "new" for malloc
    unsigned char* output = new unsigned char[output_size];

    // Copy image to host
    hipError_t err = hipMemcpy(h_conv_image, convImage, conv_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }

    // Convert float host image to unsigned char host image (0-255)
    convertToUnsignedChar(h_conv_image, output, output_size);

    // Create an OpenCV matrix for host image to use OpenCV functions
    cv::Mat convMat(outputHeight, outputHeight, CV_MAKETYPE(CV_8U, numChannels), output);

    // Save the image
    std::string outputFileName = "./output/output_" + filename + ".png";
    cv::imwrite(outputFileName, convMat);

    delete[] h_conv_image;
    delete[] output;

}


int main(int argc, char* argv[]) {

    // Initialize CUDA and cuDNN
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    auto[directory, dstWidth, dstHeight] = parseArguments(argc, argv);
    
    // Read images
    auto[h_images, srcWidth, srcHeight, numChannels, filenames] = read_images(directory);

    // Initialize convolution paramters
    int filterHeight = 5, filterWidth = 5; 
    int strideHeight = 1, strideWidth = 1;
    int paddingHeight = 1, paddingWidth = 1;
    int numFilters = 1;
    int batchSize = 2;

    // Construct the augmentor
    ImageAugmentation Augmentor(srcWidth, srcHeight, dstWidth, dstHeight, numChannels);

    // Construct the network
    CNNLayer SimpleCNN(cudnn, dstHeight, dstWidth, filterHeight, filterWidth, strideHeight, strideWidth, paddingHeight, paddingWidth, numFilters, numChannels, batchSize);

    int imageSize = numChannels * dstHeight * dstWidth;
    std::vector<float*> batch_images;
    std::vector<float> hostInput(batchSize * imageSize);
    std::vector<std::string> batch_filenames;
    for (size_t i = 0; i < h_images.size(); ++i) {
        const auto& img = h_images[i];
        const auto& filename = filenames[i]; 

        float* output = Augmentor.augment(img);

        batch_images.push_back(output);
        batch_filenames.push_back(filename);

        // When the batch is full, process it
        if (batch_images.size() == batchSize) {

            // Fill the contiguous vector with image data
            for (size_t i = 0; i < batchSize; ++i) {
                // Copy data from each individual image to the contiguous vector
                std::copy(batch_images[i], batch_images[i] + imageSize, hostInput.begin() + i * imageSize);
            }

            // Pass the batch to the network
            SimpleCNN.ForwardPass(hostInput.data()); 

            // Get the output for each image in the batch
            for (size_t j = 0; j < batch_images.size(); ++j) {
                auto[poolWidth, poolHeight, outputimage] = SimpleCNN.GetOutput(j);  
                save_image(poolWidth, poolHeight, outputimage, 1, batch_filenames[j]);
            }

            // Clear the batch
            batch_images.clear();
            hostInput.clear();
            batch_filenames.clear();
        
        }

    }

    // Cleanup
    hipdnnDestroy(cudnn);

    return 0;
}
