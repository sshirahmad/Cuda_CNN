#include <../lib/cnnlayer.h>
#include <../lib/cnn.h>
#include <../lib/augmentations.h>
#include <../lib/utils.h>
#include <random>

std::tuple<std::vector<float*>, int, int, int, std::vector<std::string>> read_images(const fs::path& directory) {
    std::vector<float*> images;
    std::vector<std::string> basenames;
    int width = 0;
    int height = 0;
    int channels = 0;

    for (const auto& entry : fs::directory_iterator(directory)) {

        if (entry.is_regular_file() && entry.path().extension() == ".png") {
            // Read the image in grayscale or unchanged
            cv::Mat img = cv::imread(entry.path().string(), cv::IMREAD_UNCHANGED);
            std::string basename = entry.path().stem().string(); 

            if (!img.empty()) {
                width = img.cols;
                height = img.rows;
                channels = img.channels();

                // Allocate memory for float image
                size_t img_size = width * height * channels;
                float* image = AllocateHostMemory<float>(img_size * sizeof(float), "pinned");

                // Convert and copy image data to float
                for (size_t i = 0; i < img_size; ++i) {
                    image[i] = static_cast<float>(img.data[i]);
                }

                images.push_back(image);
                basenames.push_back(basename);

            } else {
                std::cerr << "Failed to load image: " << entry.path() << std::endl;
            }
        } else {
            std::cerr << "Entry is not a regular file or not a PNG: " << entry.path() << std::endl;
        }
    }

    return {images, width, height, channels, basenames}; 
}



std::vector<int> readLabelsFromCSV(const std::string& fileName) {
    std::vector<int> labels;
    std::ifstream file(fileName);

    if (!file.is_open()) {
        std::cerr << "Error: Could not open the file " << fileName << std::endl;
        return labels;
    }

    std::string line;
    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string value;
        while (std::getline(ss, value, ',')) {
            try {
                labels.push_back(std::stoi(value));  // Convert the string to an integer label
            } catch (const std::invalid_argument& e) {
                std::cerr << "Invalid label: " << value << std::endl;
            }
        }
    }

    file.close();
    return labels;
}


std::tuple<std::string, int, int> parseArguments(int argc, char* argv[]) {
    // Initialize default values
    std::string directory = "../data/train/mnist_images";
    int dstWidth = 320;
    int dstHeight = 240;

    // Iterate through command-line arguments
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];

        // Check for the directory flag
        if (arg == "-d" && i + 1 < argc) {
            directory = argv[++i];
        }

        // Check for the width flag
        else if (arg == "-w" && i + 1 < argc) {
            try {
                dstWidth = std::stoi(argv[++i]);
            } catch (const std::invalid_argument& e) {
                std::cerr << "Invalid width value provided. Using default value 320." << std::endl;
            }
        }
        // Check for the height flag
        else if (arg == "-h" && i + 1 < argc) {
            try {
                dstHeight = std::stoi(argv[++i]);
            } catch (const std::invalid_argument& e) {
                std::cerr << "Invalid height value provided. Using default value 240." << std::endl;
            }
        }
    }

    std::cout << "Data Path: " << directory << std::endl;
    std::cout << "Width: " << dstWidth << std::endl;
    std::cout << "Height: " << dstHeight << std::endl;


    return {directory, dstWidth, dstHeight};
}


__host__ void convertToUnsignedChar(const float* input, unsigned char* output, int size) {

    float minRange = *std::min_element(input, input + size);
    float maxRange = *std::max_element(input, input + size);

    std::cout << minRange << std::endl;
    std::cout << maxRange << std::endl;

    if (maxRange == minRange) {
        std::fill(output, output + size, 0);  
    } else {
        for (int i = 0; i < size; ++i) {
            unsigned char scaledValue = static_cast<unsigned char>(255.0f * (input[i] - minRange) / (maxRange - minRange));
            output[i] = scaledValue;
        }
    }
}


__host__ void save_image(int outputWidth, int outputHeight, const float* convImage, int numChannels, std::string filename){

    // Calculate size of image
    int output_size = outputWidth * outputHeight * numChannels;
    size_t conv_size = output_size * sizeof(float);

    // Allocate dynamic memory to host image with flot and unsigned char types
    float* h_conv_image = new float[output_size]; // "new" for malloc
    unsigned char* output = new unsigned char[output_size];

    // Copy image to host
    hipError_t err = hipMemcpy(h_conv_image, convImage, conv_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }

    // Convert float host image to unsigned char host image (0-255)
    convertToUnsignedChar(h_conv_image, output, output_size);

    // Create an OpenCV matrix for host image to use OpenCV functions
    cv::Mat convMat(outputHeight, outputHeight, CV_MAKETYPE(CV_8U, numChannels), output);

    // Save the image
    std::string outputFileName = "./output/output_" + filename + ".png";
    cv::imwrite(outputFileName, convMat);

    delete[] h_conv_image;
    delete[] output;

}


// Function to print a progress bar
void printProgressBar(int current, int total, int barWidth = 50) {
    float progress = (float)current / total;
    int pos = barWidth * progress;

    std::cout << "[";
    for (int i = 0; i < barWidth; ++i) {
        if (i < pos) std::cout << "=";
        else if (i == pos) std::cout << ">";
        else std::cout << " ";
    }
    std::cout << "] " << int(progress * 100.0) << " %\r";
    std::cout.flush();  // Ensure the progress bar is updated in real-time
}


int main(int argc, char* argv[]) {

    // Initialize CUDA and cuDNN and cuBLAS
    hipdnnHandle_t cudnn;
    hipblasHandle_t cublas;

    hipdnnCreate(&cudnn);
    hipblasCreate(&cublas);

    auto[directory, dstWidth, dstHeight] = parseArguments(argc, argv);
    
    // Read images and labels
    std::string image_directory = directory + "mnist_images";
    std::string label_directory =  directory + "mnist_labels.csv";
    auto[h_images, srcWidth, srcHeight, numChannels, filenames] = read_images(image_directory);
    auto labels = readLabelsFromCSV(label_directory);

    // Initialize convolution paramters
    int filterHeight = 3, filterWidth = 3; 
    int strideHeight = 1, strideWidth = 1;
    int paddingHeight = 1, paddingWidth = 1;
    int numFilters = 32;
    int hiddenDim = 32, numClass = 10;
    int batchSize = 128;
    float learningrate = 0.0001;
    bool debug = false;
    int epochs = 100;

    // Construct the augmentor
    ImageAugmentation Augmentor(srcWidth, srcHeight, dstWidth, dstHeight, numChannels);

    // Construct the network
    CNN CNNModel(cudnn, cublas, srcHeight, srcWidth, filterHeight, filterWidth, strideHeight, strideWidth, paddingHeight, paddingWidth, numFilters, numChannels, hiddenDim, numClass, batchSize, learningrate);

    // Vectors for batches
    int imageSize = numChannels * dstHeight * dstWidth;
    std::vector<float> lossPerEpoch(epochs);

    std::vector<float*> batch_images;
    std::vector<float> hostInput(batchSize * imageSize);
    std::vector<int> hostLabel;
    std::vector<std::string> batch_filenames;

    for (size_t e = 0; e < epochs; ++e) {

        float epochLoss = 0.0f;

        std::cout << "Epoch " << e + 1 << "/" << epochs << std::endl;

        for (size_t i = 0; i < h_images.size(); ++i) {

            const auto& img = h_images[i];
            const auto& filename = filenames[i]; 
            const auto& label = labels[i]; 

            // Pre-process images
            float* output = Augmentor.augment(img);

            batch_images.push_back(output);
            batch_filenames.push_back(filename);
            hostLabel.push_back(label);

            // When the batch is full, process it
            if (batch_images.size() == batchSize) {

                // Fill the contiguous vector with image data
                for (size_t i = 0; i < batchSize; ++i) {
                    // Copy data from each individual image to the contiguous vector
                    std::copy(batch_images[i], batch_images[i] + imageSize, hostInput.begin() + i * imageSize);
                }

                // Pass the batch to the network
                auto logits = CNNModel.ForwardPass(hostInput.data(), hostLabel.data()); 
                auto deviceLoss = CNNModel.ComputeLoss(); 
                CNNModel.BackwardPass(); 

                // Copy batch loss from device to host
                float batchLoss = 0.0f;
                hipMemcpy(&batchLoss, deviceLoss, sizeof(float), hipMemcpyDeviceToHost);

                epochLoss += batchLoss;

                // Update the progress bar
                printProgressBar(i + 1, h_images.size());

                // Print logits for each class in each batch (debug mode)
                if (debug) {

                    std::vector<float> hostLogits(batchSize * numClass);
                    hipMemcpy(hostLogits.data(), logits, batchSize * numClass * sizeof(float), hipMemcpyDeviceToHost);

                    for (size_t i = 0; i < batchSize; ++i) {
                        std::cout << "Batch " << i << " logits: ";
                        for (size_t k = 0; k < numClass; ++k) {
                            std::cout << hostLogits[i * numClass + k] << " "; 
                        }
                        std::cout << std::endl; 
                    }

                    // Save output images
                    for (size_t j = 0; j < batch_images.size(); ++j) {
                        auto [outputWidth, outputHeight, outputImage] = CNNModel.GetOutput(j);  
                        save_image(outputWidth, outputHeight, outputImage, 1, batch_filenames[j]);
                    }
                }

                // Clear the batch
                batch_images.clear();
                hostInput.assign(batchSize * imageSize, 0.0f);  // Maintain size of hostInput
                hostLabel.clear();
                batch_filenames.clear();
            }
        }

        // Average the loss over the batches
        lossPerEpoch[e] = epochLoss / (h_images.size() / batchSize);

        // Display final loss after the epoch
        std::cout << std::endl << "Epoch " << e + 1 << " completed. Loss: " << lossPerEpoch[e] << std::endl << std::endl;
    }


    // Cleanup
    hipdnnDestroy(cudnn);
    hipblasDestroy(cublas);

    return 0;
}
