#include "../lib/augmentations.h"


ImageAugmentation::ImageAugmentation(int srcWidth, int srcHeight,
                                    int newWidth, int newHeight,
                                    int numChannels) :

                                    srcWidth(srcWidth), srcHeight(srcHeight),
                                    newWidth(newWidth), newHeight(newHeight),
                                    numChannels(numChannels)  {

                                        AllocateMemory();

                                    }


// Destructor
ImageAugmentation::~ImageAugmentation() {
    FreeMemory();
}


void ImageAugmentation::AllocateMemory(){

    hipMalloc(&deviceInput, srcWidth * srcHeight * numChannels * sizeof(float));
    hipMalloc(&deviceOutput, newWidth * newHeight * numChannels *  sizeof(float));
    h_nppOutput = nullptr;
    h_chwImage = nullptr;
}


void ImageAugmentation::FreeMemory(){

    hipFree(deviceInput);
    hipFree(deviceOutput);
    delete[] h_nppOutput;
    delete[] h_chwImage;

}


void ImageAugmentation::reset() {
    // Free previously allocated host memory
    delete[] h_nppOutput;
    // delete[] h_chwImage; // will lead to undefined behavior

    // Reset pointers to new memory
    h_nppOutput = new float[newWidth * newHeight * numChannels];
    h_chwImage = new float[newWidth * newHeight * numChannels];

    // Reset the allocated device memory
    hipMemset(deviceInput, 0, srcWidth * srcHeight * numChannels * sizeof(float));
    hipMemset(deviceOutput, 0, newWidth * newHeight * numChannels * sizeof(float));

}

float* ImageAugmentation::augment(float* h_image){

    reset();

    hipMemcpy(deviceInput, h_image, srcWidth * srcHeight * numChannels * sizeof(float), hipMemcpyHostToDevice);

    resize();
    normalize();

    // Copy NPP output from device to host (temporary storage)
    hipMemcpy(h_nppOutput, deviceOutput, newWidth * newHeight * numChannels * sizeof(float), hipMemcpyDeviceToHost);

    // Convert NPP output from HWC to CHW format
    convertHWCtoCHW(h_nppOutput, h_chwImage);

    return h_chwImage;

}


// Method to normalize the image
void ImageAugmentation::normalize() {

    NppiSize srcSize = {newWidth, newHeight};
    int srcStep = newWidth * numChannels * sizeof(float);
    NppStatus status;

    if (numChannels == 3){
        Npp32f divisor[3] = {255, 255, 255}; 
        status = nppiDivC_32f_C3IR(
            divisor, deviceOutput, srcStep, srcSize
        );

    } else if (numChannels == 4){
        Npp32f divisor[4] = {255, 255, 255, 255}; 

        status = nppiDivC_32f_C4IR(
            divisor, deviceOutput, srcStep, srcSize
        );
    

    } else if (numChannels == 1){
        Npp32f divisor = 255; 
        status = nppiDivC_32f_C1IR(
            divisor, deviceOutput, srcStep, srcSize
        );

    }

    if (status != NPP_SUCCESS) {
        throw std::runtime_error("Failed to normalize the image using NPP.");
    }

    hipDeviceSynchronize();

}

// Resizes the input image to a new width and height
void ImageAugmentation::resize() {

    NppiSize srcSize = {srcWidth, srcHeight};
    NppiRect srcRect = {0, 0, srcWidth, srcHeight};
    NppiSize dstSize = {newWidth, newHeight};
    NppiRect dstRect = {0, 0, newWidth, newHeight};
    NppStatus status;

    int srcStep = srcWidth * numChannels * sizeof(float);
    int dstStep = newWidth * numChannels * sizeof(float);

    if (numChannels == 3){

        status = nppiResize_32f_C3R(
            deviceInput, srcStep, srcSize, srcRect,
            deviceOutput, dstStep, dstSize, dstRect,
            NPPI_INTER_LINEAR
        );

    } else if (numChannels == 4){

        status = nppiResize_32f_C4R(
            deviceInput, srcStep, srcSize, srcRect,
            deviceOutput, dstStep, dstSize, dstRect,
            NPPI_INTER_LINEAR
        );
    

    } else if (numChannels == 1){

        status = nppiResize_32f_C1R(
            deviceInput, srcStep, srcSize, srcRect,
            deviceOutput, dstStep, dstSize, dstRect,
            NPPI_INTER_LINEAR
        );

    }

    if (status != NPP_SUCCESS) {
        throw std::runtime_error("Failed to resize the image using NPP.");
    }

    hipDeviceSynchronize();
    
}


// Function to convert NPP output from HWC to CHW format
void ImageAugmentation::convertHWCtoCHW(const float* srcImage, float* chwImage) {

    for (int h = 0; h < newHeight; ++h) {
        for (int w = 0; w < newWidth; ++w) {
            for (int c = 0; c < numChannels; ++c) { 
                chwImage[c * newHeight * newWidth + h * newWidth + w] = srcImage[h * newWidth * numChannels + w * numChannels + c];
            }
        }
    }
}





// Rotates the image by an angle (in degrees)
// void ImageAugmentation::rotate(const unsigned char* d_inputImage, unsigned char* d_outputImage, double angle) {
//     NppiSize srcSize = { width, height };
//     NppiRect srcRect = { 0, 0, width, height };
//     NppiSize dstSize = { width, height };
//     NppiRect dstRect = { 0, 0, width, height };

//     NppiPoint anchor = { width / 2, height / 2 };

//     int nSrcStep = width * channels * sizeof(unsigned char);
//     int nDstStep = width * channels * sizeof(unsigned char);

//     NppStatus status = nppiRotate_8u_C3R(d_inputImage, nSrcStep, srcSize, srcRect,
//                                             d_outputImage, nDstStep, dstSize, dstRect,
//                                             angle, anchor, NPPI_INTER_LINEAR);

//     if (status != NPP_SUCCESS) {
//         throw std::runtime_error("Failed to rotate the image using NPP.");
//     }
// }

// // Adjusts brightness
// void ImageAugmentation::adjustBrightness(const unsigned char* d_inputImage, unsigned char* d_outputImage, int brightness) {
//     NppiSize srcSize = { width, height };
//     int nStep = width * channels * sizeof(unsigned char);

//     NppStatus status = nppiAddC_8u_C3RSfs(d_inputImage, nStep, (Npp8u*)d_outputImage, nStep,
//                                             srcSize, brightness, 0);

//     if (status != NPP_SUCCESS) {
//         throw std::runtime_error("Failed to adjust brightness using NPP.");
//     }
// }


