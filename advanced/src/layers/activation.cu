#include "../lib/activation.h"

// CNNLayer Constructor
ActivationLayer::ActivationLayer(hipdnnHandle_t cudnn,
                    int inputHeight, int inputWidth,
                    int inputChannels, int batchSize)
                :
                cudnn(cudnn),  
                inputHeight(inputHeight), inputWidth(inputWidth),
                inputChannels(inputChannels), batchSize(batchSize) {
    
    // Initialize and set tensor and convolution descriptors
    CreateandSetDescs();
}

// Destructor
ActivationLayer::~ActivationLayer() {
    FreeMemory();
}

// Allocate memory for GPU data
void ActivationLayer::CreateandSetDescs() {

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// FORWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Input tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, inputChannels, inputHeight, inputWidth));
          
    // Activation (ReLU) descriptor
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

    // Allocate memory for activation tensor
    CHECK_CUDA(hipMalloc(&deviceOutput, batchSize * inputChannels * inputHeight * inputWidth * sizeof(float)));

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// BACKWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Allocate memory for grad of activation input tensor
    CHECK_CUDA(hipMalloc(&deviceInputGrad, batchSize * inputChannels * inputWidth * inputHeight * sizeof(float)));

}

// Free GPU memory
void ActivationLayer::FreeMemory() {

    // Clean up descriptors
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyActivationDescriptor(activationDesc);

    // Free intermediate buffers
    CHECK_CUDA(hipFree(deviceOutput));
    CHECK_CUDA(hipFree(deviceInputGrad));

}

// Forward pass
float* ActivationLayer::ForwardPass(const float* deviceInput) {

    this->deviceInput = deviceInput;

    LaunchActivationKernel();

    return deviceOutput;

}

float* ActivationLayer::BackwardPass(const float* deviceOutputGrad) {

    this->deviceOutputGrad = deviceOutputGrad;

    LaunchBackwardActivationKernel();

    return deviceInputGrad;

}

void ActivationLayer::LaunchActivationKernel() {
    float alpha = 1.0f, beta = 0.0f;

    // Apply ReLU activation function
    CHECK_CUDNN(hipdnnActivationForward(cudnn, activationDesc, &alpha, inputDesc, deviceInput,
                                       &beta, inputDesc, deviceOutput));

    hipDeviceSynchronize();

}

// Backward Activation Kernel
void ActivationLayer::LaunchBackwardActivationKernel() {
    float alpha = 1.0f, beta = 0.0f;

    CHECK_CUDNN(hipdnnActivationBackward(cudnn, activationDesc, &alpha, inputDesc, deviceOutput,
                                        inputDesc, deviceOutputGrad, inputDesc, deviceInput, &beta, inputDesc, deviceInputGrad));

    hipDeviceSynchronize();
}


