#include "hip/hip_runtime.h"
#include "../lib/cnnlayer.h"

// CNNLayer Constructor
CNNLayer::CNNLayer(hipdnnHandle_t cudnn, int inputHeight, int inputWidth,
                    int filterHeight, int filterWidth,
                    int strideHeight, int strideWidth,
                    int paddingHeight, int paddingWidth,
                    int outputChannels, int inputChannels,
                    int batchSize)
                :
    cudnn(cudnn),  
    inputHeight(inputHeight), inputWidth(inputWidth),
    filterHeight(filterHeight), filterWidth(filterWidth),
    strideHeight(strideHeight), strideWidth(strideWidth),
    paddingHeight(paddingHeight), paddingWidth(paddingWidth),
    outputChannels(outputChannels), inputChannels(inputChannels),
    batchSize(batchSize) {
    
    // Initialize and set tensor and convolution descriptors
    CreateandSetDescs();
    SetFilters();
}

// Destructor
CNNLayer::~CNNLayer() {
    FreeMemory();
}

// Allocate memory for GPU data
void CNNLayer::CreateandSetDescs() {


    // Input tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, inputChannels, inputHeight, inputWidth));
          
    // Allocate memory for pooling tensor
    hipMalloc(&deviceInput, batchSize * inputChannels * inputHeight * inputWidth * sizeof(float));

    ///////////////////////////// CONVOLUTION TENSORS AND DESCRIPTORS /////////////////////////////   
    // Filter (weights) descriptor
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                           outputChannels, inputChannels, filterHeight, filterWidth));

    // Allocate memory for filter tensor
    hipMalloc(&deviceFilter, outputChannels * inputChannels * filterHeight * filterWidth * sizeof(float));

    // Convolution descriptor
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc, paddingHeight, paddingWidth, strideHeight, strideWidth, 1, 1,
                                                HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    // Output tensor dimensions
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc,
                                                      &batchSize, &outputChannels, &convHeight, &convWidth));

    // Output tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, outputChannels, convHeight, convWidth));

    
    // Allocate memory for convolution tensor
    hipMalloc(&deviceConv, batchSize * outputChannels * convHeight * convWidth * sizeof(float));

    ///////////////////////////// ACTIVATION TENSORS AND DESCRIPTORS /////////////////////////////   

    // Activation (ReLU) descriptor
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

    ///////////////////////////// POOLING TENSORS AND DESCRIPTORS /////////////////////////////   

    // Pooling descriptor for max pooling
    CHECK_CUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    CHECK_CUDNN(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
                                            filterHeight, filterWidth, paddingHeight, paddingWidth, strideHeight, strideWidth));


    // Pooling tensor dimensions
    // poolHeight = (convHeight + 2 * paddingHeight - filterHeight) / strideHeight + 1;
    // poolWidth = (convWidth + 2 * paddingWidth - filterWidth) / strideWidth + 1;
    CHECK_CUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc, outputDesc,
                                                      &batchSize, &outputChannels, &poolHeight, &poolWidth));

    // Allocate memory for pooling tensor
    hipMalloc(&devicePool, batchSize * outputChannels * poolHeight * poolWidth * sizeof(float));


}

// Free GPU memory
void CNNLayer::FreeMemory() {

    // Clean up descriptors
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroyActivationDescriptor(activationDesc);
    hipdnnDestroyPoolingDescriptor(poolDesc);

    // Free intermediate buffers
    hipFree(deviceInput);
    hipFree(deviceConv);
    hipFree(devicePool);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Forward pass
void CNNLayer::ForwardPass(float* hostInput) {

    // reset memory
    hipMemset(deviceInput, 0, batchSize * inputWidth * inputHeight * inputChannels * sizeof(float));
    hipMemset(deviceConv, 0, batchSize * convWidth * convHeight * outputChannels * sizeof(float));
    hipMemset(devicePool, 0, batchSize * poolWidth * poolHeight * outputChannels * sizeof(float));

    // Copy the final result to the output array
    hipError_t err = hipMemcpy(deviceInput, hostInput, batchSize * inputChannels * inputHeight * inputWidth * sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                << " in File " << __FILE__
                << " in line " << __LINE__
                << std::endl;
        exit(EXIT_FAILURE);
    }

    LaunchConvolutionKernel();
    LaunchActivationKernel();
    LaunchMaxPoolingKernel();

}

void CNNLayer::LaunchConvolutionKernel() {

    // CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc,
    //                                             convDesc, outputDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, 
    //                                             &workspaceSize));

    // std::cout << workspaceSize;

    // Perform convolution
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha, inputDesc, deviceInput, filterDesc, deviceFilter,
                                        convDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, 
                                        &beta, outputDesc, deviceConv));

    hipDeviceSynchronize();

}

void CNNLayer::LaunchActivationKernel() {

    // Apply ReLU activation function
    CHECK_CUDNN(hipdnnActivationForward(cudnn, activationDesc, &alpha, outputDesc, deviceConv,
                                       &beta, outputDesc, deviceConv));

    hipDeviceSynchronize();

}

void CNNLayer::LaunchMaxPoolingKernel() {

    // Perform max pooling
    CHECK_CUDNN(hipdnnPoolingForward(cudnn, poolDesc, &alpha, outputDesc, deviceConv,
                                    &beta, outputDesc, devicePool));

    hipDeviceSynchronize();

}

// Initialize filters 
void CNNLayer::SetFilters() {
    int filter_num_elements = filterHeight * filterWidth * inputChannels * outputChannels;
    initializeWeights<<<1, filter_num_elements>>>(deviceFilter, filter_num_elements, 1234ULL, -0.5f, 0.5f);
}


// Get output from device to host
std::tuple<int, int, float*> CNNLayer::GetOutput(int index) {

    // float* output = devicePool + index * poolWidth * poolHeight * outputChannels + 0 * poolHeight * poolWidth;
    float* output = deviceConv + index * outputChannels * convHeight * convWidth + 0 * convHeight * convWidth;

    // return {poolWidth, poolHeight, output};
    return {convWidth, convHeight, output};

}
