#include "hip/hip_runtime.h"
#include "../lib/cnnlayer.h"

// CNNLayer Constructor
CNNLayer::CNNLayer(hipdnnHandle_t cudnn, int inputHeight, int inputWidth,
                    int filterHeight, int filterWidth,
                    int strideHeight, int strideWidth,
                    int paddingHeight, int paddingWidth,
                    int outputChannels, int inputChannels,
                    int batchSize, float learningrate)
                :
    cudnn(cudnn),  
    inputHeight(inputHeight), inputWidth(inputWidth),
    filterHeight(filterHeight), filterWidth(filterWidth),
    strideHeight(strideHeight), strideWidth(strideWidth),
    paddingHeight(paddingHeight), paddingWidth(paddingWidth),
    outputChannels(outputChannels), inputChannels(inputChannels),
    batchSize(batchSize), learningrate(learningrate) {
    
    // Initialize and set tensor and convolution descriptors
    CreateandSetDescs();
    SetFilters();
}

// Destructor
CNNLayer::~CNNLayer() {
    FreeMemory();
}

// Allocate memory for GPU data
void CNNLayer::CreateandSetDescs() {

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// FORWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Input tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, inputChannels, inputHeight, inputWidth));
          
    ///////////////////////////// CONVOLUTION TENSORS AND DESCRIPTORS /////////////////////////////   
    // Filter (weights) descriptor
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                           outputChannels, inputChannels, filterHeight, filterWidth));

    // Filter tensor (weights) descriptor (for hipdnnAddTensor)
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&filterTensorDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(filterTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           outputChannels, inputChannels, filterHeight, filterWidth));

    // Allocate memory for filter tensor
    CHECK_CUDA(hipMalloc(&deviceFilter, outputChannels * inputChannels * filterHeight * filterWidth * sizeof(float)));

    // Convolution descriptor
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc, paddingHeight, paddingWidth, strideHeight, strideWidth, 1, 1,
                                                HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    // Output tensor dimensions
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc,
                                                      &batchSize, &outputChannels, &convHeight, &convWidth));

    // Output convolution tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputconvDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputconvDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, outputChannels, convHeight, convWidth));

    
    // Allocate memory for convolution tensor
    CHECK_CUDA(hipMalloc(&deviceConv, batchSize * outputChannels * convHeight * convWidth * sizeof(float)));

    ///////////////////////////// ACTIVATION TENSORS AND DESCRIPTORS /////////////////////////////   

    // Activation (ReLU) descriptor
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

    // Allocate memory for activation tensor
    CHECK_CUDA(hipMalloc(&deviceAct, batchSize * outputChannels * convHeight * convWidth * sizeof(float)));

    ///////////////////////////// POOLING TENSORS AND DESCRIPTORS /////////////////////////////   

    // Pooling descriptor for max pooling
    CHECK_CUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    CHECK_CUDNN(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
                                            filterHeight, filterWidth, paddingHeight, paddingWidth, strideHeight, strideWidth));


    // Pooling tensor dimensions
    CHECK_CUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc, outputconvDesc,
                                                      &batchSize, &outputChannels, &poolHeight, &poolWidth));

    // Output pooling tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputpoolDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputpoolDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, outputChannels, poolHeight, poolWidth));

    // Allocate memory for pooling tensor
    CHECK_CUDA(hipMalloc(&deviceOutput, batchSize * outputChannels * poolHeight * poolWidth * sizeof(float)));

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// BACKWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    ///////////////////////////// POOLING TENSORS AND DESCRIPTORS /////////////////////////////   

    // Allocate memory for grad of pooling input tensor
    CHECK_CUDA(hipMalloc(&deviceActGrad, batchSize * outputChannels * convHeight * convWidth * sizeof(float)));

    ///////////////////////////// ACTIVATION TENSORS AND DESCRIPTORS /////////////////////////////   

    // Allocate memory for grad of activation input tensor
    CHECK_CUDA(hipMalloc(&deviceConvGrad, batchSize * outputChannels * convHeight * convWidth * sizeof(float)));

    ///////////////////////////// CONVOLUTION TENSORS AND DESCRIPTORS /////////////////////////////   

    // Allocate memory for grad of convolution input tensor
    CHECK_CUDA(hipMalloc(&deviceInputGrad, batchSize * inputChannels * inputWidth * inputHeight * sizeof(float)));

    // Allocate memory for grad of convolution filter tensor
    CHECK_CUDA(hipMalloc(&deviceFilterGrad, inputChannels * outputChannels * filterHeight * filterWidth * sizeof(float)));


}

// Free GPU memory
void CNNLayer::FreeMemory() {

    // Clean up descriptors
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputconvDesc);
    hipdnnDestroyTensorDescriptor(outputpoolDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroyActivationDescriptor(activationDesc);
    hipdnnDestroyPoolingDescriptor(poolDesc);

    // Free intermediate buffers
    CHECK_CUDA(hipFree(deviceConv));
    CHECK_CUDA(hipFree(deviceOutput));
    CHECK_CUDA(hipFree(deviceAct));
    CHECK_CUDA(hipFree(deviceFilter));

    CHECK_CUDA(hipFree(deviceActGrad));
    CHECK_CUDA(hipFree(deviceConvGrad));
    CHECK_CUDA(hipFree(deviceInputGrad));
    CHECK_CUDA(hipFree(deviceFilterGrad));

}

// Forward pass
float* CNNLayer::ForwardPass(const float* deviceInput) {

    this->deviceInput = deviceInput;

    LaunchConvolutionKernel();
    LaunchActivationKernel();
    LaunchMaxPoolingKernel();

    return deviceOutput;

}

float* CNNLayer::BackwardPass(const float* deviceOutputGrad) {

    this->deviceOutputGrad = deviceOutputGrad;

    LaunchBackwardMaxPoolingKernel();
    LaunchBackwardActivationKernel();
    LaunchBackwardConvolutionKernel();

    UpdateWeights();

    return deviceInputGrad;

}


void CNNLayer::UpdateWeights() {
    // Update weights (filters)
    float alpha = -learningrate; 
    float beta = 1.0f;            

    // Update filters using gradients
    CHECK_CUDNN(hipdnnAddTensor(cudnn, &alpha, filterTensorDesc, deviceFilterGrad,
                                &beta, filterTensorDesc, deviceFilter));

    hipDeviceSynchronize();
}

void CNNLayer::LaunchConvolutionKernel() {
    float alpha = 1.0f, beta = 0.0f;

    // Perform convolution
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha, inputDesc, deviceInput, filterDesc, deviceFilter,
                                        convDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, 
                                        &beta, outputconvDesc, deviceConv));

    hipDeviceSynchronize();

}

void CNNLayer::LaunchActivationKernel() {
    float alpha = 1.0f, beta = 0.0f;

    // Apply ReLU activation function
    CHECK_CUDNN(hipdnnActivationForward(cudnn, activationDesc, &alpha, outputconvDesc, deviceConv,
                                       &beta, outputconvDesc, deviceAct));

    hipDeviceSynchronize();

}

void CNNLayer::LaunchMaxPoolingKernel() {
    float alpha = 1.0f, beta = 0.0f;

    // Perform max pooling
    CHECK_CUDNN(hipdnnPoolingForward(cudnn, poolDesc, &alpha, outputconvDesc, deviceAct,
                                    &beta, outputpoolDesc, deviceOutput));

    hipDeviceSynchronize();

}

// Backward Max Pooling Kernel
void CNNLayer::LaunchBackwardMaxPoolingKernel() {

    float alpha = 1.0f, beta = 0.0f;

    CHECK_CUDNN(hipdnnPoolingBackward(cudnn, poolDesc, &alpha, outputpoolDesc, deviceOutput,
                                     outputpoolDesc, deviceOutputGrad, outputconvDesc, deviceAct, &beta, outputconvDesc, deviceActGrad));
                                     
    hipDeviceSynchronize();
}

// Backward Activation Kernel
void CNNLayer::LaunchBackwardActivationKernel() {
    float alpha = 1.0f, beta = 0.0f;

    CHECK_CUDNN(hipdnnActivationBackward(cudnn, activationDesc, &alpha, outputconvDesc, deviceAct,
                                        outputconvDesc, deviceActGrad, outputconvDesc, deviceConv, &beta, outputconvDesc, deviceConvGrad));

    hipDeviceSynchronize();
}

// Backward Convolution Kernel
void CNNLayer::LaunchBackwardConvolutionKernel() {
    float alpha = 1.0f, beta = 0.0f;

    CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn, &alpha, filterDesc, deviceFilter,
                                             outputconvDesc, deviceConvGrad, convDesc,
                                             HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                                             nullptr, 0, &beta, inputDesc, deviceInputGrad));

    CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn, &alpha, inputDesc, deviceInput,
                                               outputconvDesc, deviceConvGrad, convDesc,
                                               HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                               nullptr, 0, &beta, filterDesc, deviceFilterGrad));

    hipDeviceSynchronize();
}

// Initialize filters 
void CNNLayer::SetFilters() {

    int filter_num_elements = filterHeight * filterWidth * inputChannels * outputChannels;
    int threadsPerBlock = 256; // Choose a value that's a power of 2, usually 256 or 512
    int blocksPerGrid = (filter_num_elements + threadsPerBlock - 1) / threadsPerBlock; // Calculate total blocks needed

    initializeWeights<<<blocksPerGrid, threadsPerBlock>>>(deviceFilter, filter_num_elements, 1234ULL, -0.5f, 0.5f);

    // Ensure the kernel is executed correctly
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                << " in File " << __FILE__
                << " in line " << __LINE__
                << std::endl;
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

}


// Get output from device to host
std::tuple<int, int, float*> CNNLayer::GetOutput(int index) {

    float* output = deviceOutput + index * poolWidth * poolHeight * outputChannels + 0 * poolHeight * poolWidth;
    // float* output = deviceConv + index * outputChannels * convHeight * convWidth + 0 * convHeight * convWidth;

    return {poolWidth, poolHeight, output};
    // return {convWidth, convHeight, output};

}
