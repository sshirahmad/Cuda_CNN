#include "hip/hip_runtime.h"
#include "../lib/cnnlayer.h"

// CNNLayer Constructor
CNNLayer::CNNLayer(hipdnnHandle_t cudnn, int inputHeight, int inputWidth,
                    int filterHeight, int filterWidth,
                    int strideHeight, int strideWidth,
                    int paddingHeight, int paddingWidth,
                    int outputChannels, int inputChannels,
                    int batchSize)
                :
    cudnn(cudnn),  
    inputHeight(inputHeight), inputWidth(inputWidth),
    filterHeight(filterHeight), filterWidth(filterWidth),
    strideHeight(strideHeight), strideWidth(strideWidth),
    paddingHeight(paddingHeight), paddingWidth(paddingWidth),
    outputChannels(outputChannels), inputChannels(inputChannels),
    batchSize(batchSize) {
    
    // Initialize and set tensor and convolution descriptors
    CreateandSetDescs();
    SetFilters();
}

// Destructor
CNNLayer::~CNNLayer() {
    FreeMemory();
}

// Allocate memory for GPU data
void CNNLayer::CreateandSetDescs() {

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// FORWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Input tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, inputChannels, inputHeight, inputWidth));
          
    ///////////////////////////// CONVOLUTION TENSORS AND DESCRIPTORS /////////////////////////////   
    // Filter (weights) descriptor
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                           outputChannels, inputChannels, filterHeight, filterWidth));

    // Allocate memory for filter tensor
    hipMalloc(&deviceFilter, outputChannels * inputChannels * filterHeight * filterWidth * sizeof(float));

    // Convolution descriptor
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc, paddingHeight, paddingWidth, strideHeight, strideWidth, 1, 1,
                                                HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    // Output tensor dimensions
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc,
                                                      &batchSize, &outputChannels, &convHeight, &convWidth));

    // Output convolution tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputconvDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputconvDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, outputChannels, convHeight, convWidth));

    
    // Allocate memory for convolution tensor
    hipMalloc(&deviceConv, batchSize * outputChannels * convHeight * convWidth * sizeof(float));

    ///////////////////////////// ACTIVATION TENSORS AND DESCRIPTORS /////////////////////////////   

    // Activation (ReLU) descriptor
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

    // Allocate memory for activation tensor
    hipMalloc(&deviceAct, batchSize * outputChannels * convHeight * convWidth * sizeof(float));

    ///////////////////////////// POOLING TENSORS AND DESCRIPTORS /////////////////////////////   

    // Pooling descriptor for max pooling
    CHECK_CUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    CHECK_CUDNN(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
                                            filterHeight, filterWidth, paddingHeight, paddingWidth, strideHeight, strideWidth));


    // Pooling tensor dimensions
    CHECK_CUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc, outputconvDesc,
                                                      &batchSize, &outputChannels, &poolHeight, &poolWidth));

    // Output pooling tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputpoolDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputpoolDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, outputChannels, poolHeight, poolWidth));

    // Allocate memory for pooling tensor
    hipMalloc(&deviceOutput, batchSize * outputChannels * poolHeight * poolWidth * sizeof(float));

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// BACKWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    ///////////////////////////// POOLING TENSORS AND DESCRIPTORS /////////////////////////////   

    // Allocate memory for grad of pooling input tensor
    hipMalloc(&deviceActGrad, batchSize * outputChannels * convHeight * convWidth * sizeof(float));

    ///////////////////////////// ACTIVATION TENSORS AND DESCRIPTORS /////////////////////////////   

    // Allocate memory for grad of activation input tensor
    hipMalloc(&deviceConvGrad, batchSize * outputChannels * convHeight * convWidth * sizeof(float));

    ///////////////////////////// CONVOLUTION TENSORS AND DESCRIPTORS /////////////////////////////   

    // Allocate memory for grad of convolution input tensor
    hipMalloc(&deviceInputGrad, batchSize * inputChannels * inputWidth * inputHeight * sizeof(float));

    // Allocate memory for grad of convolution filter tensor
    hipMalloc(&deviceFilterGrad, inputChannels * outputChannels * filterHeight * filterWidth * sizeof(float));


}

// Free GPU memory
void CNNLayer::FreeMemory() {

    // Clean up descriptors
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputconvDesc);
    hipdnnDestroyTensorDescriptor(outputpoolDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroyActivationDescriptor(activationDesc);
    hipdnnDestroyPoolingDescriptor(poolDesc);

    // Free intermediate buffers
    hipFree(deviceConv);
    hipFree(deviceOutput);
    hipFree(deviceAct);
    hipFree(deviceFilter);

    hipFree(deviceActGrad);
    hipFree(deviceConvGrad);
    hipFree(deviceInputGrad);
    hipFree(deviceFilterGrad);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Forward pass
float* CNNLayer::ForwardPass(const float* Input) {

    // reset memory
    deviceInput = Input;

    hipMemset(deviceConv, 0, batchSize * convWidth * convHeight * outputChannels * sizeof(float));
    hipMemset(deviceAct, 0, batchSize * convWidth * convHeight * outputChannels * sizeof(float));
    hipMemset(deviceOutput, 0, batchSize * poolWidth * poolHeight * outputChannels * sizeof(float));

    LaunchConvolutionKernel();
    LaunchActivationKernel();
    LaunchMaxPoolingKernel();

    return deviceOutput;

}

float* CNNLayer::BackwardPass(const float* OutputGrad) {

    // Reset gradient buffers
    deviceOutputGrad = OutputGrad;

    hipMemset(deviceActGrad, 0, batchSize * outputChannels * convHeight * convWidth * sizeof(float));
    hipMemset(deviceConvGrad, 0, batchSize * outputChannels * convHeight * convWidth * sizeof(float));
    hipMemset(deviceInputGrad, 0, batchSize * inputChannels * inputHeight * inputWidth * sizeof(float));
    hipMemset(deviceFilterGrad, 0, outputChannels * inputChannels * filterHeight * filterWidth * sizeof(float));

    LaunchBackwardMaxPoolingKernel();
    LaunchBackwardActivationKernel();
    LaunchBackwardConvolutionKernel();

    return deviceInputGrad;

}


// void CNNLayer::UpdateWeights() {
//     // Update weights (filters)
//     float alpha = -learningRate; // Learning rate scaling factor for gradient descent
//     float beta = 1.0f;            // For in-place update

//     // Update filters using gradients
//     CHECK_CUDNN(hipdnnAddTensor(cudnn, &alpha, filterDesc, deviceGradFilter,
//                                 &beta, filterDesc, deviceFilter));

//     hipDeviceSynchronize();
// }

void CNNLayer::LaunchConvolutionKernel() {

    // Perform convolution
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha, inputDesc, deviceInput, filterDesc, deviceFilter,
                                        convDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, 
                                        &beta, outputconvDesc, deviceConv));

    hipDeviceSynchronize();

}

void CNNLayer::LaunchActivationKernel() {

    // Apply ReLU activation function
    CHECK_CUDNN(hipdnnActivationForward(cudnn, activationDesc, &alpha, outputconvDesc, deviceConv,
                                       &beta, outputconvDesc, deviceAct));

    hipDeviceSynchronize();

}

void CNNLayer::LaunchMaxPoolingKernel() {

    // Perform max pooling
    CHECK_CUDNN(hipdnnPoolingForward(cudnn, poolDesc, &alpha, outputconvDesc, deviceAct,
                                    &beta, outputpoolDesc, deviceOutput));

    hipDeviceSynchronize();

}

// Backward Max Pooling Kernel
void CNNLayer::LaunchBackwardMaxPoolingKernel() {
    CHECK_CUDNN(hipdnnPoolingBackward(cudnn, poolDesc, &alpha, outputpoolDesc, deviceOutput,
                                     outputpoolDesc, deviceOutputGrad, outputconvDesc, deviceAct, &beta, outputconvDesc, deviceActGrad));
                                     
    hipDeviceSynchronize();
}

// Backward Activation Kernel
void CNNLayer::LaunchBackwardActivationKernel() {
    CHECK_CUDNN(hipdnnActivationBackward(cudnn, activationDesc, &alpha, outputconvDesc, deviceAct,
                                        outputconvDesc, deviceActGrad, outputconvDesc, deviceConv, &beta, outputconvDesc, deviceConvGrad));

    hipDeviceSynchronize();
}

// Backward Convolution Kernel
void CNNLayer::LaunchBackwardConvolutionKernel() {
    CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn, &alpha, filterDesc, deviceFilter,
                                             outputconvDesc, deviceConvGrad, convDesc,
                                             HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                                             nullptr, 0, &beta, inputDesc, deviceInputGrad));

    CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn, &alpha, inputDesc, deviceInput,
                                               outputconvDesc, deviceConvGrad, convDesc,
                                               HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                               nullptr, 0, &beta, filterDesc, deviceFilterGrad));

    hipDeviceSynchronize();
}

// Initialize filters 
void CNNLayer::SetFilters() {
    int filter_num_elements = filterHeight * filterWidth * inputChannels * outputChannels;
    initializeWeights<<<1, filter_num_elements>>>(deviceFilter, filter_num_elements, 1234ULL, -0.5f, 0.5f);
}


// void CNNLayer::UpdateWeights(float learningRate) {
//     int filter_num_elements = filterHeight * filterWidth * inputChannels * outputChannels;
//     updateWeightsKernel<<<1, filter_num_elements>>>(deviceFilter, filter_num_elements, deviceFilterGrad, learningRate);
// }


// Get output from device to host
std::tuple<int, int, float*> CNNLayer::GetOutput(int index) {

    float* output = deviceOutput + index * poolWidth * poolHeight * outputChannels + 0 * poolHeight * poolWidth;
    // float* output = deviceConv + index * outputChannels * convHeight * convWidth + 0 * convHeight * convWidth;

    return {poolWidth, poolHeight, output};
    // return {convWidth, convHeight, output};

}
