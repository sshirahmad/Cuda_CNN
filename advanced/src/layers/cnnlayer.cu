#include "hip/hip_runtime.h"
#include "../lib/cnnlayer.h"

// CNNLayer Constructor
CNNLayer::CNNLayer(hipdnnHandle_t cudnn, int inputHeight, int inputWidth,
                    int filterHeight, int filterWidth,
                    int strideHeight, int strideWidth,
                    int paddingHeight, int paddingWidth,
                    int outputChannels, int inputChannels,
                    int batchSize)
                :
    cudnn(cudnn),  
    inputHeight(inputHeight), inputWidth(inputWidth),
    filterHeight(filterHeight), filterWidth(filterWidth),
    strideHeight(strideHeight), strideWidth(strideWidth),
    paddingHeight(paddingHeight), paddingWidth(paddingWidth),
    outputChannels(outputChannels), inputChannels(inputChannels),
    batchSize(batchSize) {
    
    // Initialize and set tensor and convolution descriptors
    CreateandSetDescs();
    SetFilters();
}

// Destructor
CNNLayer::~CNNLayer() {
    FreeMemory();
}

// Allocate memory for GPU data
void CNNLayer::CreateandSetDescs() {

    ///////////////////////////// FORWARD PASS /////////////////////////////   

    // Input tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, inputChannels, inputHeight, inputWidth));
          
    // Allocate memory for pooling tensor
    hipMalloc(&deviceInput, batchSize * inputChannels * inputHeight * inputWidth * sizeof(float));

    ///////////////////////////// CONVOLUTION TENSORS AND DESCRIPTORS /////////////////////////////   
    // Filter (weights) descriptor
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                           outputChannels, inputChannels, filterHeight, filterWidth));

    // Allocate memory for filter tensor
    hipMalloc(&deviceFilter, outputChannels * inputChannels * filterHeight * filterWidth * sizeof(float));

    // Convolution descriptor
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc, paddingHeight, paddingWidth, strideHeight, strideWidth, 1, 1,
                                                HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    // Output tensor dimensions
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc,
                                                      &batchSize, &outputChannels, &convHeight, &convWidth));

    // Output convolution tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputconvDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputconvDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, outputChannels, convHeight, convWidth));

    
    // Allocate memory for convolution tensor
    hipMalloc(&deviceConv, batchSize * outputChannels * convHeight * convWidth * sizeof(float));

    ///////////////////////////// ACTIVATION TENSORS AND DESCRIPTORS /////////////////////////////   

    // Activation (ReLU) descriptor
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

    ///////////////////////////// POOLING TENSORS AND DESCRIPTORS /////////////////////////////   

    // Pooling descriptor for max pooling
    CHECK_CUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    CHECK_CUDNN(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
                                            filterHeight, filterWidth, paddingHeight, paddingWidth, strideHeight, strideWidth));


    // Pooling tensor dimensions
    CHECK_CUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc, outputconvDesc,
                                                      &batchSize, &outputChannels, &poolHeight, &poolWidth));

    // Output pooling tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputpoolDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputpoolDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, outputChannels, poolHeight, poolWidth));

    // Allocate memory for pooling tensor
    hipMalloc(&devicePool, batchSize * outputChannels * poolHeight * poolWidth * sizeof(float));


    ///////////////////////////// BACKWARD PASS /////////////////////////////   



}

// Free GPU memory
void CNNLayer::FreeMemory() {

    // Clean up descriptors
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputconvDesc);
    hipdnnDestroyTensorDescriptor(outputpoolDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroyActivationDescriptor(activationDesc);
    hipdnnDestroyPoolingDescriptor(poolDesc);

    // Free intermediate buffers
    hipFree(deviceInput);
    hipFree(deviceConv);
    hipFree(devicePool);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Forward pass
void CNNLayer::ForwardPass(float* hostInput) {

    // reset memory
    hipMemset(deviceInput, 0, batchSize * inputWidth * inputHeight * inputChannels * sizeof(float));
    hipMemset(deviceConv, 0, batchSize * convWidth * convHeight * outputChannels * sizeof(float));
    hipMemset(devicePool, 0, batchSize * poolWidth * poolHeight * outputChannels * sizeof(float));

    // Copy the final result to the output array
    hipError_t err = hipMemcpy(deviceInput, hostInput, batchSize * inputChannels * inputHeight * inputWidth * sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                << " in File " << __FILE__
                << " in line " << __LINE__
                << std::endl;
        exit(EXIT_FAILURE);
    }

    LaunchConvolutionKernel();
    LaunchActivationKernel();
    LaunchMaxPoolingKernel();

}

void CNNLayer::LaunchConvolutionKernel() {

    // CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc,
    //                                             convDesc, outputDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, 
    //                                             &workspaceSize));

    // std::cout << workspaceSize;

    // Perform convolution
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha, inputDesc, deviceInput, filterDesc, deviceFilter,
                                        convDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, 
                                        &beta, outputconvDesc, deviceConv));

    hipDeviceSynchronize();

}

void CNNLayer::LaunchActivationKernel() {

    // Apply ReLU activation function
    CHECK_CUDNN(hipdnnActivationForward(cudnn, activationDesc, &alpha, outputconvDesc, deviceConv,
                                       &beta, outputconvDesc, deviceConv));

    hipDeviceSynchronize();

}

void CNNLayer::LaunchMaxPoolingKernel() {

    // Perform max pooling
    CHECK_CUDNN(hipdnnPoolingForward(cudnn, poolDesc, &alpha, outputconvDesc, deviceConv,
                                    &beta, outputpoolDesc, devicePool));

    hipDeviceSynchronize();

}

// // Backward Activation Kernel
// void CNNLayer::LaunchBackwardActivationKernel() {
//     CHECK_CUDNN(hipdnnActivationBackward(cudnn, activationDesc, &alpha, outputDesc, deviceConv,
//                                         outputDesc, deviceConv, &beta, inputDesc, deviceInput));
//     hipDeviceSynchronize();
// }

// // Backward Convolution Kernel
// void CNNLayer::LaunchBackwardConvolutionKernel(float* outputGrad) {
//     // Assuming outputGrad is the gradient from the next layer
//     CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn, &alpha, filterDesc, deviceFilter,
//                                              outputDesc, outputGrad, convDesc,
//                                              HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
//                                              nullptr, 0, &beta, inputDesc, deviceGradInput));

//     CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn, &alpha, inputDesc, deviceInput,
//                                                outputDesc, outputGrad, convDesc,
//                                                HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
//                                                nullptr, 0, &beta, filterDesc, deviceGradFilter));

//     hipDeviceSynchronize();
// }

// // Backward Max Pooling Kernel
// void CNNLayer::LaunchBackwardMaxPoolingKernel(float* outputGrad) {
//     CHECK_CUDNN(hipdnnPoolingBackward(cudnn, poolDesc, &alpha, outputDesc, devicePool,
//                                      outputDesc, outputGrad, &beta, inputDesc, deviceGradInput));
//     hipDeviceSynchronize();
// }

// Initialize filters 
void CNNLayer::SetFilters() {
    int filter_num_elements = filterHeight * filterWidth * inputChannels * outputChannels;
    initializeWeights<<<1, filter_num_elements>>>(deviceFilter, filter_num_elements, 1234ULL, -0.5f, 0.5f);
}


// Get output from device to host
std::tuple<int, int, float*> CNNLayer::GetOutput(int index) {

    float* output = devicePool + index * poolWidth * poolHeight * outputChannels + 0 * poolHeight * poolWidth;
    // float* output = deviceConv + index * outputChannels * convHeight * convWidth + 0 * convHeight * convWidth;

    return {poolWidth, poolHeight, output};
    // return {convWidth, convHeight, output};

}
