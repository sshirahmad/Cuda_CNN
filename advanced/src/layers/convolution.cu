#include "../lib/convolution.h"

// CNNLayer Constructor
ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t cudnn,
                    int inputHeight, int inputWidth,
                    int filterHeight, int filterWidth,
                    int strideHeight, int strideWidth,
                    int paddingHeight, int paddingWidth,
                    int outputChannels, int inputChannels,
                    int batchSize, float learningrate)
                :
    cudnn(cudnn),  
    inputHeight(inputHeight), inputWidth(inputWidth),
    filterHeight(filterHeight), filterWidth(filterWidth),
    strideHeight(strideHeight), strideWidth(strideWidth),
    paddingHeight(paddingHeight), paddingWidth(paddingWidth),
    outputChannels(outputChannels), inputChannels(inputChannels),
    batchSize(batchSize), learningrate(learningrate) {
    
    // Initialize and set tensor and convolution descriptors
    CreateandSetDescs();
    SetFilters();
}

// Destructor
ConvolutionLayer::~ConvolutionLayer() {
    FreeMemory();
}

// Allocate memory for GPU data
void ConvolutionLayer::CreateandSetDescs() {

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// FORWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Input tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, inputChannels, inputHeight, inputWidth));
          
    // Filter (weights) descriptor
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                           outputChannels, inputChannels, filterHeight, filterWidth));

    // Filter tensor (weights) descriptor (for hipdnnAddTensor)
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&filterTensorDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(filterTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           outputChannels, inputChannels, filterHeight, filterWidth));

    // Allocate memory for filter tensor
    CHECK_CUDA(hipMalloc(&deviceFilter, outputChannels * inputChannels * filterHeight * filterWidth * sizeof(float)));

    // Convolution descriptor
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc, paddingHeight, paddingWidth, strideHeight, strideWidth, 1, 1,
                                                HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    // Output tensor dimensions
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc,
                                                      &batchSize, &outputChannels, &convHeight, &convWidth));

    // Output convolution tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, outputChannels, convHeight, convWidth));

    
    // Allocate memory for convolution tensor
    CHECK_CUDA(hipMalloc(&deviceOutput, batchSize * outputChannels * convHeight * convWidth * sizeof(float)));

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// BACKWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Allocate memory for grad of convolution input tensor
    CHECK_CUDA(hipMalloc(&deviceInputGrad, batchSize * inputChannels * inputWidth * inputHeight * sizeof(float)));

    // Allocate memory for grad of convolution filter tensor
    CHECK_CUDA(hipMalloc(&deviceFilterGrad, inputChannels * outputChannels * filterHeight * filterWidth * sizeof(float)));


}

// Free GPU memory
void ConvolutionLayer::FreeMemory() {

    // Clean up descriptors
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);

    // Free intermediate buffers
    CHECK_CUDA(hipFree(deviceOutput));
    CHECK_CUDA(hipFree(deviceFilter));
    CHECK_CUDA(hipFree(deviceInputGrad));
    CHECK_CUDA(hipFree(deviceFilterGrad));

}

// Forward pass
float* ConvolutionLayer::ForwardPass(const float* deviceInput) {

    this->deviceInput = deviceInput;

    LaunchConvolutionKernel();

    return deviceOutput;

}

float* ConvolutionLayer::BackwardPass(const float* deviceOutputGrad) {

    this->deviceOutputGrad = deviceOutputGrad;

    LaunchBackwardConvolutionKernel();

    UpdateWeights();

    return deviceInputGrad;

}

void ConvolutionLayer::UpdateWeights() {
    // Update weights (filters)
    float alpha = -learningrate; 
    float beta = 1.0f;            

    // Update filters using gradients
    CHECK_CUDNN(hipdnnAddTensor(cudnn, &alpha, filterTensorDesc, deviceFilterGrad,
                                &beta, filterTensorDesc, deviceFilter));

    hipDeviceSynchronize();
}

void ConvolutionLayer::LaunchConvolutionKernel() {
    float alpha = 1.0f, beta = 0.0f;

    // Perform convolution
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha, inputDesc, deviceInput, filterDesc, deviceFilter,
                                        convDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, 
                                        &beta, outputDesc, deviceOutput));

    hipDeviceSynchronize();

}

// Backward Convolution Kernel
void ConvolutionLayer::LaunchBackwardConvolutionKernel() {
    float alpha = 1.0f, beta = 0.0f;

    CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn, &alpha, filterDesc, deviceFilter,
                                             outputDesc, deviceOutputGrad, convDesc,
                                             HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                                             nullptr, 0, &beta, inputDesc, deviceInputGrad));

    CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn, &alpha, inputDesc, deviceInput,
                                               outputDesc, deviceOutputGrad, convDesc,
                                               HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                               nullptr, 0, &beta, filterDesc, deviceFilterGrad));

    hipDeviceSynchronize();
}

// Initialize filters 
void ConvolutionLayer::SetFilters() {

    int filter_num_elements = filterHeight * filterWidth * inputChannels * outputChannels;
    int threadsPerBlock = 256; // Choose a value that's a power of 2, usually 256 or 512
    int blocksPerGrid = (filter_num_elements + threadsPerBlock - 1) / threadsPerBlock; // Calculate total blocks needed

    // initializeUniformWeights<<<blocksPerGrid, threadsPerBlock>>>(deviceFilter, filter_num_elements, 1234ULL, -0.0f, 0.01f);
    initializeXavierWeights<<<blocksPerGrid, threadsPerBlock>>>(deviceFilter, filter_num_elements, 1234ULL, filterHeight * filterWidth * inputChannels);

    // Ensure the kernel is executed correctly
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                << " in File " << __FILE__
                << " in line " << __LINE__
                << std::endl;
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

}

