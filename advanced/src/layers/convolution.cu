#include "../lib/convolution.h"

// CNNLayer Constructor
ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t cudnn, hipblasHandle_t cublas,
                    int inputHeight, int inputWidth,
                    int filterHeight, int filterWidth,
                    int strideHeight, int strideWidth,
                    int paddingHeight, int paddingWidth,
                    int outputChannels, int inputChannels,
                    int batchSize, float learningrate,
                    float weight_decay)
                :
    cudnn(cudnn), cublas(cublas),
    inputHeight(inputHeight), inputWidth(inputWidth),
    filterHeight(filterHeight), filterWidth(filterWidth),
    strideHeight(strideHeight), strideWidth(strideWidth),
    paddingHeight(paddingHeight), paddingWidth(paddingWidth),
    outputChannels(outputChannels), inputChannels(inputChannels),
    batchSize(batchSize), learningrate(learningrate),
    weight_decay(weight_decay){
    
    // Initialize and set tensor and convolution descriptors
    CreateandSetDescs();
    SetFilters();
}

// Destructor
ConvolutionLayer::~ConvolutionLayer() {
    FreeMemory();
}

// Allocate memory for GPU data
void ConvolutionLayer::CreateandSetDescs() {

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// FORWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Input tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, inputChannels, inputHeight, inputWidth));
          
    // Filter (weights) descriptor
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                           outputChannels, inputChannels, filterHeight, filterWidth));

    // Filter tensor (weights) descriptor (for hipdnnAddTensor)
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&filterTensorDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(filterTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           outputChannels, inputChannels, filterHeight, filterWidth));

    // Allocate memory for filter tensor
    CHECK_CUDA(hipMalloc(&deviceFilter, outputChannels * inputChannels * filterHeight * filterWidth * sizeof(float)));

    // Convolution descriptor
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc, paddingHeight, paddingWidth, strideHeight, strideWidth, 1, 1,
                                                HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    // Output tensor dimensions
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc,
                                                      &batchSize, &outputChannels, &convHeight, &convWidth));

    // Output convolution tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, outputChannels, convHeight, convWidth));

    
    // Allocate memory for convolution tensor
    CHECK_CUDA(hipMalloc(&deviceOutput, batchSize * outputChannels * convHeight * convWidth * sizeof(float)));

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// BACKWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Allocate memory for grad of convolution input tensor
    CHECK_CUDA(hipMalloc(&deviceInputGrad, batchSize * inputChannels * inputWidth * inputHeight * sizeof(float)));

    // Allocate memory for grad of convolution filter tensor
    CHECK_CUDA(hipMalloc(&deviceFilterGrad, inputChannels * outputChannels * filterHeight * filterWidth * sizeof(float)));

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// OPTIMIZERS ////////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    optimizer = new Adam(inputChannels * outputChannels * filterHeight * filterWidth, batchSize, learningrate, weight_decay);


}

// Free GPU memory
void ConvolutionLayer::FreeMemory() {

    // Clean up descriptors
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);

    // Free intermediate buffers
    CHECK_CUDA(hipFree(deviceOutput));
    CHECK_CUDA(hipFree(deviceFilter));
    CHECK_CUDA(hipFree(deviceInputGrad));
    CHECK_CUDA(hipFree(deviceFilterGrad));

    delete optimizer;

}

// Forward pass
float* ConvolutionLayer::ForwardPass(const float* deviceInput) {

    this->deviceInput = deviceInput;

    LaunchConvolutionKernel();

    return deviceOutput;

}

float* ConvolutionLayer::BackwardPass(const float* deviceOutputGrad) {

    this->deviceOutputGrad = deviceOutputGrad;

    LaunchBackwardConvolutionKernel();

    UpdateWeights();

    return deviceInputGrad;

}


void ConvolutionLayer::UpdateWeights() {

    optimizer->update(deviceFilter, deviceFilterGrad);

    hipDeviceSynchronize();

}

void ConvolutionLayer::LaunchConvolutionKernel() {
    float alpha = 1.0f, beta = 0.0f;

    // Perform convolution
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha, inputDesc, deviceInput, filterDesc, deviceFilter,
                                        convDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, 
                                        &beta, outputDesc, deviceOutput));

    hipDeviceSynchronize();

}

// Backward Convolution Kernel
void ConvolutionLayer::LaunchBackwardConvolutionKernel() {
    float alpha = 1.0f, beta = 0.0f;

    CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn, &alpha, filterDesc, deviceFilter,
                                             outputDesc, deviceOutputGrad, convDesc,
                                             HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                                             nullptr, 0, &beta, inputDesc, deviceInputGrad));

    CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn, &alpha, inputDesc, deviceInput,
                                               outputDesc, deviceOutputGrad, convDesc,
                                               HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                               nullptr, 0, &beta, filterDesc, deviceFilterGrad));

    hipDeviceSynchronize();
}

// Initialize filters 
void ConvolutionLayer::SetFilters() {

    int filter_num_elements = filterHeight * filterWidth * inputChannels * outputChannels;
    int threadsPerBlock = 256; // Choose a value that's a power of 2, usually 256 or 512
    int blocksPerGrid = (filter_num_elements + threadsPerBlock - 1) / threadsPerBlock; // Calculate total blocks needed

    initializeUniformWeights<<<blocksPerGrid, threadsPerBlock>>>(deviceFilter, filter_num_elements, 1234ULL, 0.0f, 0.01f);
    // initializeXavierWeights<<<blocksPerGrid, threadsPerBlock>>>(deviceFilter, filter_num_elements, 1234ULL, filterHeight * filterWidth * inputChannels);

    // Ensure the kernel is executed correctly
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                << " in File " << __FILE__
                << " in line " << __LINE__
                << std::endl;
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

}


void ConvolutionLayer::SaveWeights(FILE* file) {

    if (!file) {
        std::cerr << "Invalid file pointer for loading convolutional weights." << std::endl;
        return;
    }

    // Save layer dimensions (input size, output size)
    fwrite(&inputChannels, sizeof(int), 1, file);
    fwrite(&outputChannels, sizeof(int), 1, file);
    fwrite(&filterHeight, sizeof(int), 1, file);
    fwrite(&filterWidth, sizeof(int), 1, file);

    // Determine the size of the weights
    int filter_num_elements = outputChannels * inputChannels * filterHeight * filterWidth;
    
    // Allocate memory on the host (CPU) to store the weights
    std::vector<float> hostFilter(filter_num_elements);

    // Copy weights from device (GPU) to host (CPU)
    CHECK_CUDA(hipMemcpy(hostFilter.data(), deviceFilter, filter_num_elements * sizeof(float), hipMemcpyDeviceToHost));

    // Write the weights to the file
    fwrite(hostFilter.data(), sizeof(float), filter_num_elements, file);

}



void ConvolutionLayer::LoadWeights(FILE* file) {

    if (!file) {
        std::cerr << "Invalid file pointer for loading convolutional weights." << std::endl;
        return;
    }


    // Read layer dimensions (input size, output size)
    int loadedInputChannels, loadedOutputChannels, loadedFilterHeight, loadedFilterWidth;
    fread(&loadedInputChannels, sizeof(int), 1, file);
    fread(&loadedOutputChannels, sizeof(int), 1, file);
    fread(&loadedFilterHeight, sizeof(int), 1, file);
    fread(&loadedFilterWidth, sizeof(int), 1, file);

    // Ensure that dimensions match
    if (loadedInputChannels != inputChannels || loadedOutputChannels != outputChannels || loadedFilterHeight != filterHeight || loadedFilterWidth != filterWidth) {
        std::cerr << "Layer dimensions mismatch while loading weights and biases!" << std::endl;
        fclose(file);
        return;
    }

    // Determine the size of the weights
    int filter_num_elements = outputChannels * inputChannels * filterHeight * filterWidth;

    // Allocate memory on the host (CPU) to store the weights
    std::vector<float> hostFilter(filter_num_elements);

    // Read the weights from the file
    fread(hostFilter.data(), sizeof(float), filter_num_elements, file);

    // Copy the weights from host (CPU) to device (GPU)
    CHECK_CUDA(hipMemcpy(deviceFilter, hostFilter.data(), filter_num_elements * sizeof(float), hipMemcpyHostToDevice));

}


