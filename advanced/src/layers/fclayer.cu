#include "../lib/fclayer.h"

// Constructor
FCLayer::FCLayer(hipblasHandle_t cublasHandle, int inputSize, int outputSize, int batchSize, float learningRate, float weight_decay)
    : cublasHandle(cublasHandle), inputSize(inputSize), outputSize(outputSize), batchSize(batchSize), learningRate(learningRate), weight_decay(weight_decay) {
    AllocateMemory();
    InitializeWeights();
}

// Destructor
FCLayer::~FCLayer() {
    FreeMemory();
}

// Allocate device memory
void FCLayer::AllocateMemory() {
    CHECK_CUDA(hipMalloc(&deviceWeight, inputSize * outputSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&deviceBias, outputSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&deviceOutput, batchSize * outputSize * sizeof(float)));

    CHECK_CUDA(hipMalloc(&deviceInputGrad, batchSize * inputSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&deviceWeightGrad, inputSize * outputSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&deviceBiasGrad, outputSize * sizeof(float)));

    CHECK_CUDA(hipMalloc(&ones, batchSize * sizeof(float)));

    CHECK_CUDA(hipMemset(ones, 1.0f, batchSize * sizeof(float)));

    optimizer_weights = new Adam(inputSize * outputSize, batchSize, learningRate, weight_decay);
    optimizer_bias = new Adam(outputSize, batchSize, learningRate);


}

// Free device memory
void FCLayer::FreeMemory() {
    CHECK_CUDA(hipFree(deviceWeight));
    CHECK_CUDA(hipFree(deviceBias));
    CHECK_CUDA(hipFree(deviceOutput));
    CHECK_CUDA(hipFree(deviceInputGrad));
    CHECK_CUDA(hipFree(deviceWeightGrad));
    CHECK_CUDA(hipFree(deviceBiasGrad));
    CHECK_CUDA(hipFree(ones));

    delete optimizer_weights;
    delete optimizer_bias;

}

// Forward pass
float* FCLayer::ForwardPass(const float* deviceInput) {

    const float alpha = 1.0f;
    const float beta = 0.0f;

    this->deviceInput = deviceInput;

    // Forward pass using cuBLAS
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, 
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                outputSize, batchSize, inputSize,  // Note: outputSize x batchSize x inputSize
                &alpha, 
                deviceWeight, outputSize,          // deviceWeight should be of shape (inputSize, outputSize)
                this->deviceInput, inputSize,      // deviceInput (cnnOutput) is of shape (batchSize, inputSize)
                &beta, 
                deviceOutput, outputSize));   

    // Add biases
    CHECK_CUBLAS(hipblasSger(cublasHandle,
            outputSize, batchSize,        
            &alpha,
            deviceBias, 1,                      
            ones, 1,                 
            deviceOutput, outputSize));

    
    hipDeviceSynchronize();


    return deviceOutput; 
}

// Backward pass
float* FCLayer::BackwardPass(const float* deviceOutputGrad) {

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Assign the deviceOutputGrad pointer
    this->deviceOutputGrad = deviceOutputGrad;

    // Gradient w.r.t input
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, 
                    HIPBLAS_OP_T, HIPBLAS_OP_N,
                    inputSize, batchSize, outputSize,  // Adjusted: inputSize x batchSize x outputSize
                    &alpha, 
                    deviceWeight, outputSize,            // deviceWeight: shape (inputSize, outputSize)
                    this->deviceOutputGrad, outputSize, // deviceOutputGrad: shape (batchSize, outputSize)
                    &beta, 
                    deviceInputGrad, inputSize));        // deviceInputGrad: shape (inputSize, batchSize)

    // Compute gradients with respect to weights (dW)
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, 
                    HIPBLAS_OP_N, HIPBLAS_OP_T,
                    outputSize, inputSize, batchSize,   // Adjusted: outputSize x inputSize x batchSize
                    &alpha, 
                    this->deviceOutputGrad, outputSize,  // deviceOutputGrad: shape (batchSize, outputSize)
                    deviceInput, inputSize,               // deviceInput: shape (batchSize, inputSize)
                    &beta, 
                    deviceWeightGrad, outputSize));       // deviceWeightGrad: shape (inputSize, outputSize)

    // Gradient w.r.t biases (db)
    CHECK_CUBLAS(hipblasSgemv(cublasHandle, 
                    HIPBLAS_OP_N, 
                    outputSize, batchSize,                // Correct dimensions: outputSize x batchSize
                    &alpha, 
                    this->deviceOutputGrad, outputSize,   // deviceOutputGrad: shape (batchSize, outputSize)
                    ones, 1,                               // `ones`: vector of size (batchSize)
                    &beta, 
                    deviceBiasGrad, 1));                  // deviceBiasGrad: shape (outputSize)

    
    hipDeviceSynchronize();

    // Update weights and biases
    UpdateWeightsAndBiases();

    return deviceInputGrad; 
}

void FCLayer::UpdateWeightsAndBiases() {

    optimizer_weights->update(deviceWeight, deviceWeightGrad);
    optimizer_bias->update(deviceBias, deviceBiasGrad);

    hipDeviceSynchronize();

}


// Initialize weights and biases
void FCLayer::InitializeWeights() {

    int weight_num_elements = inputSize * outputSize;
    int threadsPerBlock = 256; // Choose a value that's a power of 2, usually 256 or 512
    int blocksPerGrid = (weight_num_elements + threadsPerBlock - 1) / threadsPerBlock; // Calculate total blocks needed

    initializeUniformWeights<<<blocksPerGrid, threadsPerBlock>>>(deviceWeight, weight_num_elements, 1234ULL, 0.0f, 0.01f);
    // initializeXavierWeights<<<blocksPerGrid, threadsPerBlock>>>(deviceWeight, weight_num_elements, 1234ULL, inputSize);

    int bias_num_elements = outputSize;
    blocksPerGrid = (bias_num_elements + threadsPerBlock - 1) / threadsPerBlock; // Calculate total blocks needed

    initializeBias<<<blocksPerGrid, threadsPerBlock>>>(deviceBias, bias_num_elements, 0.0f);


    // Ensure the kernel is executed correctly
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                << " in File " << __FILE__
                << " in line " << __LINE__
                << std::endl;
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

}


void FCLayer::SaveWeightsAndBiases(FILE* file) {

    if (!file) {
        std::cerr << "Invalid file pointer for loading convolutional weights." << std::endl;
        return;
    }

    // Save layer dimensions (input size, output size)
    fwrite(&inputSize, sizeof(int), 1, file);
    fwrite(&outputSize, sizeof(int), 1, file);

    // Allocate host memory for weights and biases
    std::vector<float> hostWeights(inputSize * outputSize);
    std::vector<float> hostBias(outputSize);

    // Copy data from device (GPU) to host (CPU)
    CHECK_CUDA(hipMemcpy(hostWeights.data(), deviceWeight, inputSize * outputSize * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(hostBias.data(), deviceBias, outputSize * sizeof(float), hipMemcpyDeviceToHost));

    // Write weights to file
    fwrite(hostWeights.data(), sizeof(float), inputSize * outputSize, file);

    // Write biases to file
    fwrite(hostBias.data(), sizeof(float), outputSize, file);

}


void FCLayer::LoadWeightsAndBiases(FILE* file) {

    if (!file) {
        std::cerr << "Invalid file pointer for loading convolutional weights." << std::endl;
        return;
    }

    // Read layer dimensions (input size, output size)
    int loadedInputSize, loadedOutputSize;
    fread(&loadedInputSize, sizeof(int), 1, file);
    fread(&loadedOutputSize, sizeof(int), 1, file);

    // Ensure that dimensions match
    if (loadedInputSize != inputSize || loadedOutputSize != outputSize) {
        std::cerr << "Layer dimensions mismatch while loading weights and biases!" << std::endl;
        fclose(file);
        return;
    }

    // Allocate host memory for weights and biases
    std::vector<float> hostWeights(inputSize * outputSize);
    std::vector<float> hostBias(outputSize);

    // Read weights from file
    fread(hostWeights.data(), sizeof(float), inputSize * outputSize, file);

    // Read biases from file
    fread(hostBias.data(), sizeof(float), outputSize, file);

    // Copy data from host (CPU) to device (GPU)
    CHECK_CUDA(hipMemcpy(deviceWeight, hostWeights.data(), inputSize * outputSize * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(deviceBias, hostBias.data(), outputSize * sizeof(float), hipMemcpyHostToDevice));

}
