#include "hip/hip_runtime.h"
#include "../lib/kernel.h"


__global__ void initializeWeights(float* weights, int size, unsigned long long seed, float min, float max) {
    
    // Define the CUDA random state
    hiprandState state;
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Initialize the random state with the seed
    hiprand_init(seed, idx, 0, &state);

    // Make sure to not go out of bounds
    if (idx < size) {
        // Generate a random float in the range [min, max]
        float randValue = hiprand_uniform(&state) * (max - min) + min;
        weights[idx] = randValue;
    }
}



__global__ void flatten_NCHW(float* input, float* output, int batchSize, int channels, int height, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int sizePerImage = channels * height * width;

    if (idx < batchSize * sizePerImage) {
        int batchIdx = idx / sizePerImage;
        int innerIdx = idx % sizePerImage;

        // Flatten: output stores each image as a row
        output[batchIdx * sizePerImage + innerIdx] = input[idx];
    }
}
