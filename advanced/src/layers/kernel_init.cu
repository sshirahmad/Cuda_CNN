#include "hip/hip_runtime.h"
#include "../lib/kernel_init.h"


__global__ void initializeWeights(float* weights, int size, unsigned long long seed, float min, float max) {
    
    // Define the CUDA random state
    hiprandState state;
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Initialize the random state with the seed
    hiprand_init(seed, idx, 0, &state);

    // Make sure to not go out of bounds
    if (idx < size) {
        // Generate a random float in the range [min, max]
        float randValue = hiprand_uniform(&state) * (max - min) + min;
        weights[idx] = randValue;
    }
}