#include "hip/hip_runtime.h"
#include "../lib/pooling.h"

// CNNLayer Constructor
PoolingLayer::PoolingLayer(hipdnnHandle_t cudnn,
                            int inputHeight, int inputWidth,
                            int filterHeight, int filterWidth,
                            int strideHeight, int strideWidth,
                            int paddingHeight, int paddingWidth,
                            int inputChannels, int batchSize)
                :
    cudnn(cudnn),  
    inputHeight(inputHeight), inputWidth(inputWidth),
    filterHeight(filterHeight), filterWidth(filterWidth),
    strideHeight(strideHeight), strideWidth(strideWidth),
    paddingHeight(paddingHeight), paddingWidth(paddingWidth),
    inputChannels(inputChannels), batchSize(batchSize) {
    
    // Initialize and set tensor and convolution descriptors
    CreateandSetDescs();

}

// Destructor
PoolingLayer::~PoolingLayer() {
    FreeMemory();
}

// Allocate memory for GPU data
void PoolingLayer::CreateandSetDescs() {

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// FORWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Input tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, inputChannels, inputHeight, inputWidth));

    // Pooling descriptor for max pooling
    CHECK_CUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    CHECK_CUDNN(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
                                            filterHeight, filterWidth, paddingHeight, paddingWidth, strideHeight, strideWidth));


    // Pooling tensor dimensions
    CHECK_CUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc, inputDesc,
                                                  &batchSize, &inputChannels, &poolHeight, &poolWidth));

    // Output pooling tensor descriptor
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                           batchSize, inputChannels, poolHeight, poolWidth));

    // Allocate memory for pooling tensor
    CHECK_CUDA(hipMalloc(&deviceOutput, batchSize * inputChannels * poolHeight * poolWidth * sizeof(float)));

    /////////////////////////////////////////////////////////////////////////
    ///////////////////////////// BACKWARD PASS /////////////////////////////   
    /////////////////////////////////////////////////////////////////////////

    // Allocate memory for grad of pooling input tensor
    CHECK_CUDA(hipMalloc(&deviceInputGrad, batchSize * inputChannels * inputWidth * inputHeight * sizeof(float)));

}

// Free GPU memory
void PoolingLayer::FreeMemory() {

    // Clean up descriptors
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
    hipdnnDestroyPoolingDescriptor(poolDesc);

    // Free intermediate buffers
    CHECK_CUDA(hipFree(deviceOutput));
    CHECK_CUDA(hipFree(deviceInputGrad));
}

// Forward pass
float* PoolingLayer::ForwardPass(const float* deviceInput) {

    this->deviceInput = deviceInput;

    LaunchMaxPoolingKernel();

    return deviceOutput;

}

float* PoolingLayer::BackwardPass(const float* deviceOutputGrad) {

    this->deviceOutputGrad = deviceOutputGrad;

    LaunchBackwardMaxPoolingKernel();

    return deviceInputGrad;

}

void PoolingLayer::LaunchMaxPoolingKernel() {
    float alpha = 1.0f, beta = 0.0f;

    // Perform max pooling
    CHECK_CUDNN(hipdnnPoolingForward(cudnn, poolDesc, &alpha, inputDesc, deviceInput,
                                    &beta, outputDesc, deviceOutput));

    hipDeviceSynchronize();

}

// Backward Max Pooling Kernel
void PoolingLayer::LaunchBackwardMaxPoolingKernel() {

    float alpha = 1.0f, beta = 0.0f;

    CHECK_CUDNN(hipdnnPoolingBackward(cudnn, poolDesc, &alpha, outputDesc, deviceOutput,
                                     outputDesc, deviceOutputGrad, inputDesc, deviceInput, &beta, inputDesc, deviceInputGrad));
                                     
    hipDeviceSynchronize();
}

